#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <array>
#include "job_upload_api.h"

#include "cxxsha1.hpp"

extern "C" __global__
void sha1_double_kernel(uint8_t *, uint64_t *, uint32_t *, uint64_t);

#define CUDA_CHECK(x)                                                     \
    do { hipError_t e = (x);                                             \
         if (e != hipSuccess) {                                          \
             std::cerr << hipGetErrorString(e) << '\n'; std::exit(1);}   \
    } while (0)

/* --------------------------------------------------------------------- */
constexpr uint32_t BATCH = 1u << 22; // 4 194 304 threads/launch
constexpr int THREADS = 1024;
constexpr uint32_t RING_SIZE = 1u << 20; // ring buffer slots

int main(int argc, char **argv) {
    int repeats = 50;
    if (argc > 1) {
        char *end = nullptr;
        long tmp = std::strtol(argv[1], &end, 10);
        if (end != argv[1] && *end == '\0' && tmp > 0) repeats = int(tmp);
    }

    /* ------------------------------------------------------------------ */
    /* 1.  Challenge pre-image and its double-SHA-1 digest                */
    /* ------------------------------------------------------------------ */
    std::array<uint8_t, 32> preimage = {
        /* TODO: insert the 32-byte puzzle here */
        0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07,
        0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f,
        0x10, 0x11, 0x12, 0x13, 0x14, 0x15, 0x16, 0x17,
        0x18, 0x19, 0x1a, 0x1b, 0x1c, 0x1d, 0x1e, 0x1f
    };

    uint8_t d1[20], d2[20];
    sha1_ctx c{};
    sha1_init(c);
    sha1_update(c, preimage.data(), 32);
    sha1_final(c, d1);
    sha1_init(c);
    sha1_update(c, d1, 20); // second SHA-1
    sha1_final(c, d2);

    uint32_t target[5];
    for (int i = 0; i < 5; ++i) {
        target[i] =
                (static_cast<uint32_t>(d2[4 * i]) << 24) |
                (static_cast<uint32_t>(d2[4 * i + 1]) << 16) |
                (static_cast<uint32_t>(d2[4 * i + 2]) << 8) |
                static_cast<uint32_t>(d2[4 * i + 3]);
    }

    upload_new_job(preimage.data(), target);

    /* ------------------------------------------------------------------ */
    /* 2.  Allocate ring buffer + ticket                                  */
    /* ------------------------------------------------------------------ */
    uint64_t *d_pairs = nullptr;
    uint32_t *d_ticket = nullptr;
    CUDA_CHECK(hipMalloc(&d_pairs, sizeof(uint64_t) * 6 * RING_SIZE));
    CUDA_CHECK(hipMalloc(&d_ticket, sizeof(uint32_t)));
    CUDA_CHECK(hipMemset(d_ticket, 0, sizeof(uint32_t)));

    constexpr dim3 blockDim(THREADS);
    constexpr dim3 gridDim((BATCH + THREADS - 1) / THREADS);

    /* ------------------------------------------------------------------ */
    /* 3.  Timing loop                                                    */
    /* ------------------------------------------------------------------ */
    hipEvent_t start{}, stop{};
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    for (int i = 0; i < repeats; ++i) {
        sha1_double_kernel<<<gridDim, blockDim>>>(
            nullptr, d_pairs, d_ticket, 0xCAFEBABEULL + i);
        CUDA_CHECK(hipPeekAtLastError());
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float ms = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    /* ------------------------------------------------------------------ */
    /* 4.  Statistics                                                     */
    /* ------------------------------------------------------------------ */
    double hashes = double(gridDim.x) * blockDim.x * repeats;
    double seconds = ms / 1e3;
    double ghps = hashes / seconds / 1e9;

    std::cout.setf(std::ios::fixed);
    std::cout.precision(2);
    std::cout << "Kernel time   : " << seconds << " s\n"
            << "Hash-rate     : " << ghps * 1000.0
            << " M double-SHA-1/s\n"
            << "PCIe traffic  : 0.00 GiB/s (all data stays on-GPU)\n";

    uint32_t found = 0;
    CUDA_CHECK(hipMemcpy(&found, d_ticket, sizeof(found),
        hipMemcpyDeviceToHost));
    std::cout << "Candidates stored: " << found << '\n';

    /* ------------------------------------------------------------------ */
    /* 5.  Cleanup                                                        */
    /* ------------------------------------------------------------------ */
    CUDA_CHECK(hipFree(d_pairs));
    CUDA_CHECK(hipFree(d_ticket));
    return 0;
}
