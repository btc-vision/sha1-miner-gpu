#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <array>
#include <vector>
#include <chrono>
#include <thread>
#include <atomic>
#include <csignal>
#include <fstream>
#include <sstream>
#include <ctime>
#include <mutex>
#include <memory>
#include <algorithm>
#include <random>
#include <cstring>
#include <deque>

#ifdef _WIN32
#include <windows.h>
#else
    #include <pthread.h>
    #include <sched.h>
#endif

#include "job_upload_api.h"
#include "cxxsha1.hpp"

// Kernel declarations - updated to match our fixed kernels
extern "C" __global__ void sha1_mining_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_warp_collaborative_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_vectorized_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_bitsliced_kernel_correct(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_hashcat_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_hashcat_extreme_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_cooperative_groups_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_multi_hash_kernel(uint64_t *, uint32_t *, uint64_t); // good

extern "C" __global__ void sha1_readonly_cache_kernel(uint64_t *, uint32_t *, uint64_t); // bad

extern "C" __global__ void sha1_simd_vectorized_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_hybrid_warp_simd_kernel(uint64_t *, uint32_t *, uint64_t); // very good?

extern "C" __global__ void sha1_ldg_optimized_kernel(uint64_t *, uint32_t *, uint64_t); // bad

#define CUDA_CHECK(e) do{ hipError_t _e=(e); \
    if(_e!=hipSuccess){ \
        std::cerr << "CUDA Error: " << hipGetErrorString(_e) \
                  << " at " << __FILE__ << ":" << __LINE__ << '\n'; \
        std::exit(1);} \
    }while(0)

// ==================== Configuration ====================
constexpr uint32_t RING_SIZE = 1u << 20; // 1M candidate slots
constexpr int PROGRESS_INTERVAL = 5; // Update every 5 seconds
constexpr int STREAMS_PER_GPU = 8; // More streams for better overlap

// Kernel types
enum KernelType {
    KERNEL_STANDARD,
    KERNEL_WARP_COLLABORATIVE,
    KERNEL_VECTORIZED,
    KERNEL_BITSLICED,
    KERNEL_HASHCAT,
    KERNEL_HASHCAT_EXTREME,
    KERNEL_COOPERATIVE_GROUPS,
    KERNEL_MULTI_HASH,
    KERNEL_READONLY_CACHE,
    KERNEL_SIMD_VECTORIZED,
    KERNEL_HYBRID_WARP_SIMD,
    KERNEL_LDG_OPTIMIZED
};

// Global state for signal handling
std::atomic<bool> g_shutdown(false);
std::atomic<uint64_t> g_total_hashes(0);
std::atomic<uint32_t> g_total_candidates(0);
std::atomic<uint64_t> g_total_kernel_launches(0);
std::chrono::steady_clock::time_point g_start_time;

// Signal handler for graceful shutdown
void signal_handler(int sig) {
    std::cout << "\n\nReceived signal " << sig << ", shutting down gracefully...\n";
    g_shutdown.store(true);
}

void setup_signal_handlers() {
    std::signal(SIGINT, signal_handler);
    std::signal(SIGTERM, signal_handler);
#ifdef _WIN32
    std::signal(SIGBREAK, signal_handler);
#endif
}

// Helper to get kernel name
const char *getKernelName(KernelType type) {
    static const char *names[] = {
        "Standard", "Warp Collaborative", "Vectorized", "Bitsliced",
        "HashCat", "HashCat Extreme", "Cooperative Groups", "Multi-Hash",
        "Read-Only Cache", "SIMD Vectorized", "Hybrid Warp-SIMD", "LDG Optimized"
    };
    return names[type];
}

// ==================== GPU Configuration ====================
struct GPUContext {
    int device_id;
    hipDeviceProp_t properties;
    std::vector<hipStream_t> streams;
    std::vector<uint64_t *> d_pairs;
    std::vector<uint32_t *> d_tickets;
    int optimal_blocks;
    int optimal_threads;
    std::atomic<uint64_t> hashes_processed;
    std::atomic<uint64_t> kernels_launched;
    KernelType kernel_type;
    int warps_per_block;
    uint64_t hashes_per_kernel_launch;

    GPUContext(int id, bool force_bitsliced = false, bool use_hashcat = true,
               int force_kernel_type = -1) : device_id(id), hashes_processed(0), kernels_launched(0) {
        CUDA_CHECK(hipSetDevice(device_id));
        CUDA_CHECK(hipGetDeviceProperties(&properties, device_id));

        // Determine optimal configuration based on GPU and kernel type
        if (force_kernel_type >= 0 && force_kernel_type <= 11) {
            kernel_type = (KernelType) force_kernel_type;

            // Set configuration based on actual kernel implementation
            switch (kernel_type) {
                case KERNEL_STANDARD:
                    optimal_threads = 256;
                    warps_per_block = optimal_threads / 32;
                    optimal_blocks = properties.multiProcessorCount * 8;
                    // sha1_mining_kernel: NONCES_PER_THREAD = 4
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 4;
                    break;

                case KERNEL_WARP_COLLABORATIVE:
                    optimal_threads = 256;
                    warps_per_block = 8;
                    optimal_blocks = properties.multiProcessorCount * 32;
                    // sha1_warp_collaborative_kernel: BATCHES_PER_WARP = 8, 32 hashes per batch
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 8 * 32;
                    break;

                case KERNEL_VECTORIZED:
                    optimal_threads = 128;
                    warps_per_block = optimal_threads / 32;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_vectorized_kernel: 2 nonces per thread
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 2;
                    break;

                case KERNEL_BITSLICED:
                    optimal_threads = 128;
                    warps_per_block = 4;
                    optimal_blocks = properties.multiProcessorCount * 8;
                    // sha1_bitsliced_kernel_correct: BATCHES = 4, 32 messages per batch per warp
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 4 * 32;
                    break;

                case KERNEL_HASHCAT:
                    optimal_threads = 64;
                    warps_per_block = 2;
                    optimal_blocks = properties.multiProcessorCount * 64;
                    // sha1_hashcat_kernel: HASHES_PER_THREAD = 16
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 16;
                    break;

                case KERNEL_HASHCAT_EXTREME:
                    optimal_threads = 128;
                    warps_per_block = 4;
                    optimal_blocks = properties.multiProcessorCount * 32;
                    // sha1_hashcat_extreme_kernel: HASHES_PER_THREAD = 32
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 32;
                    break;

                case KERNEL_COOPERATIVE_GROUPS:
                    optimal_threads = 256;
                    warps_per_block = 8;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_cooperative_groups_kernel: BATCHES = 10, 32 hashes per batch per warp
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 10 * 32;
                    break;

                case KERNEL_MULTI_HASH:
                    optimal_threads = 128;
                    warps_per_block = optimal_threads / 32;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_multi_hash_kernel: HASHES_PER_THREAD = 8
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 8;
                    break;

                case KERNEL_READONLY_CACHE:
                    optimal_threads = 256;
                    warps_per_block = optimal_threads / 32;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_readonly_cache_kernel: NONCES_PER_THREAD = 4
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 4;
                    break;

                case KERNEL_SIMD_VECTORIZED:
                    optimal_threads = 256;
                    warps_per_block = optimal_threads / 32;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_simd_vectorized_kernel: VECTORS_PER_THREAD = 2, 4 hashes per vector
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 2 * 4;
                    break;

                case KERNEL_HYBRID_WARP_SIMD:
                    optimal_threads = 128;
                    warps_per_block = 4;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_hybrid_warp_simd_kernel: BATCHES = 4, HASHES_PER_THREAD = 2, 32 threads per warp
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 4 * 32 * 2;
                    break;

                case KERNEL_LDG_OPTIMIZED:
                    optimal_threads = 256;
                    warps_per_block = optimal_threads / 32;
                    optimal_blocks = properties.multiProcessorCount * 16;
                    // sha1_ldg_optimized_kernel: NONCES_PER_THREAD = 6
                    hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 6;
                    break;
            }
        } else if (force_bitsliced) {
            kernel_type = KERNEL_BITSLICED;
            optimal_threads = 128;
            warps_per_block = 4;
            optimal_blocks = properties.multiProcessorCount * 8;
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 4 * 32;
        } else if (use_hashcat) {
            if (properties.major >= 8) {
                kernel_type = KERNEL_HASHCAT_EXTREME;
                optimal_threads = 128;
                warps_per_block = 4;
                optimal_blocks = properties.multiProcessorCount * 32;
                hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 32;
            } else {
                kernel_type = KERNEL_HASHCAT;
                optimal_threads = 64;
                warps_per_block = 2;
                optimal_blocks = properties.multiProcessorCount * 64;
                hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 16;
            }
        } else if (properties.major >= 7) {
            kernel_type = KERNEL_WARP_COLLABORATIVE;
            optimal_threads = 256;
            warps_per_block = 8;
            optimal_blocks = properties.multiProcessorCount * 32;
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 8 * 32;
        } else if (properties.major >= 6) {
            kernel_type = KERNEL_VECTORIZED;
            optimal_threads = 128;
            warps_per_block = 4;
            optimal_blocks = properties.multiProcessorCount * 16;
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 2;
        } else {
            kernel_type = KERNEL_STANDARD;
            optimal_threads = 256;
            warps_per_block = 8;
            optimal_blocks = properties.multiProcessorCount * 8;
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 4;
        }

        // Check available memory
        size_t free_mem, total_mem;
        CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

        // Reserve some memory for system
        free_mem = (size_t) (free_mem * 0.9);

        size_t mem_per_stream = sizeof(uint64_t) * 4 * RING_SIZE + sizeof(uint32_t);
        int num_streams = std::min(STREAMS_PER_GPU, (int) (free_mem / mem_per_stream));

        // Create streams and allocate memory
        streams.resize(num_streams);
        d_pairs.resize(num_streams);
        d_tickets.resize(num_streams);

        for (int i = 0; i < num_streams; i++) {
            CUDA_CHECK(hipStreamCreateWithPriority(&streams[i], hipStreamNonBlocking, i % 2));
            CUDA_CHECK(hipMalloc(&d_pairs[i], sizeof(uint64_t) * 4 * RING_SIZE));
            CUDA_CHECK(hipMalloc(&d_tickets[i], sizeof(uint32_t)));
            CUDA_CHECK(hipMemsetAsync(d_tickets[i], 0, sizeof(uint32_t), streams[i]));
        }

        // Set cache configuration based on kernel
        switch (kernel_type) {
            case KERNEL_STANDARD:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_mining_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_WARP_COLLABORATIVE:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_warp_collaborative_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_VECTORIZED:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_vectorized_kernel), hipFuncCachePreferShared));
                break;
            case KERNEL_BITSLICED:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_bitsliced_kernel_correct), hipFuncCachePreferL1));
                break;
            case KERNEL_HASHCAT:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_hashcat_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_HASHCAT_EXTREME:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_hashcat_extreme_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_COOPERATIVE_GROUPS:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_cooperative_groups_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_MULTI_HASH:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_multi_hash_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_READONLY_CACHE:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_readonly_cache_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_SIMD_VECTORIZED:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_simd_vectorized_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_HYBRID_WARP_SIMD:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_hybrid_warp_simd_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_LDG_OPTIMIZED:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_ldg_optimized_kernel), hipFuncCachePreferL1));
                break;
        }

        printInfo();
    }

    ~GPUContext() {
        CUDA_CHECK(hipSetDevice(device_id));
        for (size_t i = 0; i < streams.size(); i++) {
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            CUDA_CHECK(hipStreamDestroy(streams[i]));
            CUDA_CHECK(hipFree(d_pairs[i]));
            CUDA_CHECK(hipFree(d_tickets[i]));
        }
    }

    void printInfo() {
        std::cout << "=== GPU " << device_id << " Configuration ===\n";
        std::cout << "Device        : " << properties.name << "\n";
        std::cout << "Compute Cap   : " << properties.major << "." << properties.minor << "\n";
        std::cout << "SMs           : " << properties.multiProcessorCount << "\n";
        std::cout << "Clock Rate    : " << properties.clockRate / 1000 << " MHz\n";
        std::cout << "Memory        : " << properties.totalGlobalMem / (1024.0 * 1024 * 1024) << " GB\n";
        std::cout << "Memory Clock  : " << properties.memoryClockRate / 1000 << " MHz\n";
        std::cout << "Memory Bus    : " << properties.memoryBusWidth << " bits\n";
        std::cout << "L2 Cache      : " << properties.l2CacheSize / (1024 * 1024) << " MB\n";

        std::cout << "Kernel        : " << getKernelName(kernel_type) << "\n";
        std::cout << "Configuration : " << optimal_blocks << " blocks × " << optimal_threads << " threads\n";

        // Display kernel-specific information
        switch (kernel_type) {
            case KERNEL_STANDARD:
            case KERNEL_READONLY_CACHE:
                std::cout << "Work per thread: 4 hashes\n";
                break;
            case KERNEL_WARP_COLLABORATIVE:
                std::cout << "Work per warp: 8 batches × 32 hashes = 256 hashes\n";
                break;
            case KERNEL_VECTORIZED:
                std::cout << "Work per thread: 2 hashes\n";
                break;
            case KERNEL_BITSLICED:
                std::cout << "Work per warp: 4 batches × 32 messages = 128 hashes\n";
                break;
            case KERNEL_HASHCAT:
                std::cout << "Work per thread: 16 hashes\n";
                break;
            case KERNEL_HASHCAT_EXTREME:
                std::cout << "Work per thread: 32 hashes\n";
                break;
            case KERNEL_COOPERATIVE_GROUPS:
                std::cout << "Work per warp: 10 batches × 32 hashes = 320 hashes\n";
                break;
            case KERNEL_MULTI_HASH:
                std::cout << "Work per thread: 8 hashes\n";
                break;
            case KERNEL_SIMD_VECTORIZED:
                std::cout << "Work per thread: 2 vectors × 4 hashes = 8 hashes\n";
                break;
            case KERNEL_HYBRID_WARP_SIMD:
                std::cout << "Work per warp: 4 batches × 64 hashes = 256 hashes\n";
                break;
            case KERNEL_LDG_OPTIMIZED:
                std::cout << "Work per thread: 6 hashes\n";
                break;
        }

        std::cout << "Hashes/Launch : " << hashes_per_kernel_launch << "\n";
        std::cout << "Streams       : " << streams.size() << "\n\n";
    }

    void launchKernel(hipStream_t stream, uint64_t seed) {
        int stream_idx = getStreamIndex(stream);

        switch (kernel_type) {
            case KERNEL_STANDARD:
                sha1_mining_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_WARP_COLLABORATIVE:
                sha1_warp_collaborative_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_VECTORIZED:
                sha1_vectorized_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_BITSLICED:
                sha1_bitsliced_kernel_correct<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_HASHCAT:
                sha1_hashcat_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_HASHCAT_EXTREME:
                sha1_hashcat_extreme_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_COOPERATIVE_GROUPS:
                sha1_cooperative_groups_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_MULTI_HASH:
                sha1_multi_hash_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_READONLY_CACHE:
                sha1_readonly_cache_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_SIMD_VECTORIZED:
                sha1_simd_vectorized_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_HYBRID_WARP_SIMD:
                sha1_hybrid_warp_simd_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
            case KERNEL_LDG_OPTIMIZED:
                sha1_ldg_optimized_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[stream_idx], d_tickets[stream_idx], seed);
                break;
        }

        kernels_launched++;
        g_total_kernel_launches.fetch_add(1);
    }

private:
    int getStreamIndex(hipStream_t stream) {
        for (size_t i = 0; i < streams.size(); i++) {
            if (streams[i] == stream) return i;
        }
        return 0;
    }
};

// ==================== Result Handler ====================
class ResultHandler {
private:
    std::ofstream output_file;
    std::mutex file_mutex;
    std::array<uint8_t, 20> target_hash;
    std::array<uint8_t, 32> base_message;

public:
    ResultHandler(const std::string &filename, const uint8_t target[20], const uint8_t base_msg[32]) {
        std::copy(target, target + 20, target_hash.begin());
        std::copy(base_msg, base_msg + 32, base_message.begin());

        output_file.open(filename, std::ios::app);
        if (!output_file.is_open()) {
            std::cerr << "Failed to open output file: " << filename << "\n";
        }

        auto now = std::chrono::system_clock::now();
        auto time_t = std::chrono::system_clock::to_time_t(now);
        output_file << "\n=== SHA-1 Collision Mining Started at "
                << std::put_time(std::localtime(&time_t), "%Y-%m-%d %H:%M:%S")
                << " ===\n";
        output_file << "Target: ";
        for (int i = 0; i < 20; i++) {
            output_file << std::hex << std::setw(2) << std::setfill('0') << (int) target_hash[i];
        }
        output_file << "\n\n";
        output_file.flush();
    }

    void saveResults(uint64_t *h_pairs, uint32_t count, int gpu_id) {
        std::lock_guard<std::mutex> lock(file_mutex);

        output_file << "[GPU " << gpu_id << "] Found " << count << " collision candidates:\n";

        for (uint32_t i = 0; i < std::min(count, 100u); i++) {
            output_file << "Candidate " << i + 1 << ":\n";

            // Reconstruct message
            uint8_t msg[32];
            for (int j = 0; j < 4; j++) {
                uint64_t word = h_pairs[i * 4 + j];
                for (int k = 0; k < 8; k++) {
                    msg[j * 8 + k] = (word >> (k * 8)) & 0xFF;
                }
            }

            output_file << "  Message: ";
            for (int j = 0; j < 32; j++) {
                output_file << std::hex << std::setw(2) << std::setfill('0') << (int) msg[j];
            }
            output_file << "\n";

            // Verify SHA-1
            uint8_t hash[20];
            sha1_ctx ctx;
            sha1_init(ctx);
            sha1_update(ctx, msg, 32);
            sha1_final(ctx, hash);

            output_file << "  SHA-1:   ";
            for (int j = 0; j < 20; j++) {
                output_file << std::hex << std::setw(2) << std::setfill('0') << (int) hash[j];
            }

            bool verified = (std::memcmp(hash, target_hash.data(), 20) == 0);
            output_file << " [" << (verified ? "VERIFIED" : "FAILED") << "]\n\n";
        }

        output_file.flush();

        // Console notification
        std::cout << "\n[GPU " << gpu_id << "] Found " << count << " collision candidates! ";
        std::cout << "Saved to output file.\n";
    }
};

// ==================== Performance Monitor ====================
class PerformanceMonitor {
private:
    std::chrono::steady_clock::time_point last_update;
    uint64_t last_hashes = 0;
    uint64_t last_kernel_launches = 0;
    std::mutex monitor_mutex;
    std::vector<double> rate_history;
    const size_t history_size = 12; // 1 minute of 5-second intervals

public:
    PerformanceMonitor() {
        last_update = std::chrono::steady_clock::now();
        rate_history.reserve(history_size);
    }

    void update(const std::vector<std::unique_ptr<GPUContext> > &gpus) {
        auto now = std::chrono::steady_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - last_update);

        if (elapsed.count() >= PROGRESS_INTERVAL) {
            std::lock_guard<std::mutex> lock(monitor_mutex);

            auto total_elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - g_start_time);
            double total_seconds = total_elapsed.count();

            uint64_t current_hashes = g_total_hashes.load();
            uint64_t current_kernel_launches = g_total_kernel_launches.load();
            uint64_t interval_hashes = current_hashes - last_hashes;
            uint64_t interval_kernel_launches = current_kernel_launches - last_kernel_launches;

            double interval_rate = interval_hashes / elapsed.count() / 1e9;
            double average_rate = current_hashes / total_seconds / 1e9;
            double kernel_rate = interval_kernel_launches / (double) elapsed.count();

            // Update rate history
            rate_history.push_back(interval_rate);
            if (rate_history.size() > history_size) {
                rate_history.erase(rate_history.begin());
            }

            // Calculate moving average
            double moving_avg = 0;
            for (double rate: rate_history) {
                moving_avg += rate;
            }
            moving_avg /= rate_history.size();

            std::cout << "\r[" << formatTime(total_seconds) << "] "
                    << "Hashes: " << std::fixed << std::setprecision(2)
                    << current_hashes / 1e12 << "T | "
                    << "Rate: " << interval_rate << " GH/s (avg: "
                    << average_rate << " GH/s, MA: " << moving_avg << ") | "
                    << "Kernels/s: " << std::fixed << std::setprecision(0) << kernel_rate << " | "
                    << "Candidates: " << g_total_candidates.load();

            // Per-GPU stats
            std::cout << " | GPU: ";
            for (size_t i = 0; i < gpus.size(); i++) {
                double gpu_rate = gpus[i]->hashes_processed.load() / total_seconds / 1e9;
                std::cout << "[" << i << ":" << std::fixed << std::setprecision(1)
                        << gpu_rate << "] ";
            }
            std::cout << "    " << std::flush;

            last_update = now;
            last_hashes = current_hashes;
            last_kernel_launches = current_kernel_launches;
        }
    }

    void finalReport() {
        auto now = std::chrono::steady_clock::now();
        auto total_elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - g_start_time);
        double total_seconds = total_elapsed.count();
        uint64_t total_hashes = g_total_hashes.load();
        uint64_t total_kernel_launches = g_total_kernel_launches.load();
        double average_rate = total_hashes / total_seconds / 1e9;

        std::cout << "\n\n=== Final Statistics ===\n";
        std::cout << "Total Time    : " << formatTime(total_seconds) << "\n";
        std::cout << "Total Hashes  : " << std::fixed << std::setprecision(3)
                << total_hashes / 1e12 << " trillion\n";
        std::cout << "Average Rate  : " << average_rate << " GH/s\n";
        std::cout << "Kernel Launches: " << total_kernel_launches << "\n";
        std::cout << "Avg Kernel Time: " << std::fixed << std::setprecision(2)
                << (total_seconds * 1000.0) / total_kernel_launches << " ms\n";
        std::cout << "Candidates    : " << g_total_candidates.load() << "\n";
        std::cout << "Efficiency    : " << std::scientific << std::setprecision(2)
                << (double) g_total_candidates.load() / total_hashes * 100 << "%\n";
    }

private:
    std::string formatTime(double seconds) {
        int hours = seconds / 3600;
        int minutes = (seconds - hours * 3600) / 60;
        int secs = seconds - hours * 3600 - minutes * 60;

        std::stringstream ss;
        ss << std::setfill('0') << std::setw(2) << hours << ":"
                << std::setw(2) << minutes << ":"
                << std::setw(2) << secs;
        return ss.str();
    }
};

// ==================== GPU Worker Thread ====================
void gpu_worker(GPUContext *gpu, ResultHandler *results, uint64_t base_seed) {
    CUDA_CHECK(hipSetDevice(gpu->device_id));

    uint64_t local_seed = base_seed + gpu->device_id * (1ull << 48);

    // Allocate pinned host memory for faster transfers
    std::vector<uint64_t> h_pairs;
    h_pairs.resize(RING_SIZE * 4);

    // Create events for accurate timing
    std::vector<hipEvent_t> start_events(gpu->streams.size());
    std::vector<hipEvent_t> stop_events(gpu->streams.size());
    std::vector<bool> kernel_running(gpu->streams.size(), false);
    std::vector<uint64_t> kernel_seeds(gpu->streams.size());

    for (size_t i = 0; i < gpu->streams.size(); i++) {
        CUDA_CHECK(hipEventCreate(&start_events[i]));
        CUDA_CHECK(hipEventCreate(&stop_events[i]));
    }

    // Keep track of kernels launched for debugging
    uint64_t local_kernels_launched = 0;
    auto last_debug_time = std::chrono::steady_clock::now();

    while (!g_shutdown.load()) {
        // Launch kernels on available streams
        for (size_t s = 0; s < gpu->streams.size(); s++) {
            if (!kernel_running[s]) {
                // Record start time
                CUDA_CHECK(hipEventRecord(start_events[s], gpu->streams[s]));

                // Launch kernel
                gpu->launchKernel(gpu->streams[s], local_seed);
                kernel_seeds[s] = local_seed;
                local_seed += gpu->hashes_per_kernel_launch;

                // Record stop time
                CUDA_CHECK(hipEventRecord(stop_events[s], gpu->streams[s]));

                kernel_running[s] = true;
                local_kernels_launched++;
            }
        }

        // Check completed kernels
        for (size_t s = 0; s < gpu->streams.size(); s++) {
            if (kernel_running[s]) {
                hipError_t err = hipEventQuery(stop_events[s]);
                if (err == hipSuccess) {
                    // Kernel completed
                    kernel_running[s] = false;

                    // Check for results
                    uint32_t found = 0;
                    CUDA_CHECK(hipMemcpyAsync(&found, gpu->d_tickets[s], sizeof(uint32_t),
                        hipMemcpyDeviceToHost, gpu->streams[s]));
                    CUDA_CHECK(hipStreamSynchronize(gpu->streams[s]));

                    if (found > 0) {
                        g_total_candidates.fetch_add(found);

                        uint32_t to_copy = std::min(found, RING_SIZE);
                        CUDA_CHECK(hipMemcpyAsync(h_pairs.data(), gpu->d_pairs[s],
                            sizeof(uint64_t) * 4 * to_copy, hipMemcpyDeviceToHost, gpu->streams[s]));
                        CUDA_CHECK(hipStreamSynchronize(gpu->streams[s]));

                        results->saveResults(h_pairs.data(), to_copy, gpu->device_id);

                        // Reset ticket
                        CUDA_CHECK(hipMemsetAsync(gpu->d_tickets[s], 0, sizeof(uint32_t), gpu->streams[s]));
                    }

                    // Update hash count
                    g_total_hashes.fetch_add(gpu->hashes_per_kernel_launch);
                    gpu->hashes_processed.fetch_add(gpu->hashes_per_kernel_launch);
                } else if (err != hipErrorNotReady) {
                    CUDA_CHECK(err);
                }
            }
        }

        // Periodic debug output (every 30 seconds)
        auto now = std::chrono::steady_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - last_debug_time);
        if (elapsed.count() >= 30 && gpu->device_id == 0) {
            double kernels_per_sec = local_kernels_launched / (double) elapsed.count();
            std::cout << "\n[GPU 0 Debug] Kernels: " << local_kernels_launched
                    << " in " << elapsed.count() << "s = "
                    << std::fixed << std::setprecision(1) << kernels_per_sec
                    << " kernels/sec, Expected: "
                    << (kernels_per_sec * gpu->hashes_per_kernel_launch / 1e9)
                    << " GH/s\n" << std::flush;
            local_kernels_launched = 0;
            last_debug_time = now;
        }

        // Small yield to prevent CPU spinning
        std::this_thread::yield();
    }

    // Cleanup
    for (auto &stream: gpu->streams) {
        CUDA_CHECK(hipStreamSynchronize(stream));
    }

    for (size_t i = 0; i < gpu->streams.size(); i++) {
        CUDA_CHECK(hipEventDestroy(start_events[i]));
        CUDA_CHECK(hipEventDestroy(stop_events[i]));
    }
}

// ==================== Benchmark Utilities ====================
void benchmarkKernel(GPUContext *gpu, int warmup_runs = 10, int test_runs = 100) {
    CUDA_CHECK(hipSetDevice(gpu->device_id));

    std::cout << "\n=== Benchmarking Kernel Performance ===\n";
    std::cout << "Kernel: " << getKernelName(gpu->kernel_type) << "\n";
    std::cout << "Configuration: " << gpu->optimal_blocks << " blocks × "
            << gpu->optimal_threads << " threads\n";
    std::cout << "Hashes per launch: " << gpu->hashes_per_kernel_launch << "\n\n";

    // Warmup
    for (int i = 0; i < warmup_runs; i++) {
        gpu->launchKernel(gpu->streams[0], i * gpu->hashes_per_kernel_launch);
    }
    CUDA_CHECK(hipStreamSynchronize(gpu->streams[0]));

    // Create events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Time the kernel
    CUDA_CHECK(hipEventRecord(start, gpu->streams[0]));

    for (int i = 0; i < test_runs; i++) {
        gpu->launchKernel(gpu->streams[0], i * gpu->hashes_per_kernel_launch);
    }

    CUDA_CHECK(hipEventRecord(stop, gpu->streams[0]));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    double seconds = milliseconds / 1000.0;
    uint64_t total_hashes = (uint64_t) test_runs * gpu->hashes_per_kernel_launch;
    double ghps = total_hashes / seconds / 1e9;

    std::cout << "Results:\n";
    std::cout << "  Total time: " << milliseconds << " ms\n";
    std::cout << "  Time per kernel: " << milliseconds / test_runs << " ms\n";
    std::cout << "  Total hashes: " << total_hashes << "\n";
    std::cout << "  Performance: " << ghps << " GH/s\n";
    std::cout << "  Theoretical max (all streams): " << ghps * gpu->streams.size()
            << " GH/s\n\n";

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
}

// ==================== Configuration Parser ====================
struct Config {
    std::vector<int> gpu_ids;
    std::string output_file = "sha1_collisions.txt";
    std::array<uint8_t, 32> target_preimage;
    bool benchmark_mode = false;
    int benchmark_seconds = 60;
    bool force_bitsliced = false;
    bool no_hashcat = false;
    int force_kernel = -1; // -1 = auto, 0-11 = specific kernel
    bool quick_bench = false;
};

Config parse_arguments(int argc, char **argv) {
    Config config;

    // Initialize with zeros
    config.target_preimage.fill(0);

    for (int i = 1; i < argc; i++) {
        std::string arg = argv[i];

        if (arg == "--gpu" && i + 1 < argc) {
            config.gpu_ids.push_back(std::stoi(argv[++i]));
        } else if (arg == "--output" && i + 1 < argc) {
            config.output_file = argv[++i];
        } else if (arg == "--target" && i + 1 < argc) {
            std::string hex = argv[++i];
            if (hex.length() != 64) {
                std::cerr << "Error: Target must be 64 hex characters (32 bytes)\n";
                std::exit(1);
            }
            for (int j = 0; j < 32; j++) {
                std::string byte = hex.substr(j * 2, 2);
                config.target_preimage[j] = std::stoi(byte, nullptr, 16);
            }
        } else if (arg == "--benchmark") {
            config.benchmark_mode = true;
            if (i + 1 < argc && argv[i + 1][0] != '-') {
                config.benchmark_seconds = std::stoi(argv[++i]);
            }
        } else if (arg == "--quick-bench") {
            config.quick_bench = true;
        } else if (arg == "--bitsliced") {
            config.force_bitsliced = true;
        } else if (arg == "--no-hashcat") {
            config.no_hashcat = true;
        } else if (arg == "--kernel" && i + 1 < argc) {
            config.force_kernel = std::stoi(argv[++i]);
        } else if (arg == "--list-kernels") {
            std::cout << "Available kernels:\n";
            std::cout << "  0: Standard (4 hashes/thread)\n";
            std::cout << "  1: Warp Collaborative (256 hashes/warp) - FASTEST on modern GPUs\n";
            std::cout << "  2: Vectorized (2 hashes/thread)\n";
            std::cout << "  3: Bitsliced (128 hashes/warp)\n";
            std::cout << "  4: HashCat (16 hashes/thread)\n";
            std::cout << "  5: HashCat Extreme (32 hashes/thread) - Best for Ampere+\n";
            std::cout << "  6: Cooperative Groups (320 hashes/warp)\n";
            std::cout << "  7: Multi-Hash (8 hashes/thread)\n";
            std::cout << "  8: Read-Only Cache (4 hashes/thread)\n";
            std::cout << "  9: SIMD Vectorized (8 hashes/thread)\n";
            std::cout << " 10: Hybrid Warp-SIMD (256 hashes/warp)\n";
            std::cout << " 11: LDG Optimized (6 hashes/thread)\n";
            std::exit(0);
        } else if (arg == "--help") {
            std::cout << "SHA-1 Collision Miner v3.0\n\n";
            std::cout << "Usage: " << argv[0] << " [options]\n\n";
            std::cout << "Options:\n";
            std::cout << "  --gpu <id>       GPU device ID (can be used multiple times)\n";
            std::cout << "  --output <file>  Output file (default: sha1_collisions.txt)\n";
            std::cout << "  --target <hex>   Target preimage in hex (default: zeros)\n";
            std::cout << "  --benchmark [s]  Run benchmark for s seconds (default: 60)\n";
            std::cout << "  --quick-bench    Run quick kernel benchmark\n";
            std::cout << "  --bitsliced      Force bitsliced kernel (slower, for testing)\n";
            std::cout << "  --no-hashcat     Use original kernels instead of HashCat-style\n";
            std::cout << "  --kernel <id>    Force specific kernel (0-11, see --list-kernels)\n";
            std::cout << "  --list-kernels   List all available kernels\n";
            std::cout << "  --help           Show this help\n\n";
            std::cout << "Example:\n";
            std::cout << "  " << argv[0] << " --gpu 0 --gpu 1 --target ";
            std::cout << "0123456789abcdef0123456789abcdef0123456789abcdef0123456789abcdef\n";
            std::exit(0);
        } else {
            std::cerr << "Unknown option: " << arg << "\n";
            std::cerr << "Use --help for usage information\n";
            std::exit(1);
        }
    }

    // Default: use all available GPUs
    if (config.gpu_ids.empty()) {
        int num_gpus;
        CUDA_CHECK(hipGetDeviceCount(&num_gpus));
        for (int i = 0; i < num_gpus; i++) {
            config.gpu_ids.push_back(i);
        }
    }

    return config;
}

// ==================== Main Function ====================
int main(int argc, char **argv) {
    setup_signal_handlers();

    std::cout << "\n+------------------------------------------+\n";
    std::cout << "|      SHA-1 Collision Miner v3.0         |\n";
    std::cout << "+------------------------------------------+\n\n";

    // Parse configuration
    Config config = parse_arguments(argc, argv);

    // Calculate target SHA-1
    uint8_t target_hash[20];
    sha1_ctx ctx;
    sha1_init(ctx);
    sha1_update(ctx, config.target_preimage.data(), 32);
    sha1_final(ctx, target_hash);

    // Convert to uint32_t array for GPU
    uint32_t target[5];
    for (int i = 0; i < 5; i++) {
        target[i] = (uint32_t(target_hash[4 * i]) << 24) |
                    (uint32_t(target_hash[4 * i + 1]) << 16) |
                    (uint32_t(target_hash[4 * i + 2]) << 8) |
                    uint32_t(target_hash[4 * i + 3]);
    }

    // Upload job to GPU constant memory
    upload_new_job(config.target_preimage.data(), target);

    std::cout << "Target Preimage: ";
    for (int i = 0; i < 32; i++) {
        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int) config.target_preimage[i];
    }
    std::cout << "\n";

    std::cout << "Target SHA-1: ";
    for (int i = 0; i < 20; i++) {
        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int) target_hash[i];
    }
    std::cout << std::dec << "\n\n";

    if (config.force_bitsliced) {
        std::cout << "NOTE: Using bitsliced kernel (slower, for testing)\n\n";
    } else if (config.force_kernel >= 0) {
        std::cout << "NOTE: Forcing kernel " << config.force_kernel << " ("
                << getKernelName((KernelType) config.force_kernel) << ")\n\n";
    }

    // Initialize GPUs
    std::vector<std::unique_ptr<GPUContext> > gpus;
    for (int id: config.gpu_ids) {
        try {
            gpus.push_back(std::make_unique<GPUContext>(
                id, config.force_bitsliced, !config.no_hashcat, config.force_kernel));
        } catch (const std::exception &e) {
            std::cerr << "Failed to initialize GPU " << id << ": " << e.what() << "\n";
        }
    }

    if (gpus.empty()) {
        std::cerr << "No GPUs available for mining!\n";
        return 1;
    }

    // Run quick benchmark if requested
    if (config.quick_bench) {
        for (auto &gpu: gpus) {
            benchmarkKernel(gpu.get(), 10, 100);
        }
        return 0;
    }

    // Initialize result handler
    ResultHandler results(config.output_file, target_hash, config.target_preimage.data());

    // Initialize performance monitor
    PerformanceMonitor monitor;

    // Start timing
    g_start_time = std::chrono::steady_clock::now();

    // Generate random base seed
    std::random_device rd;
    uint64_t base_seed = (uint64_t(rd()) << 32) | rd();

    std::cout << "Starting collision search with " << gpus.size() << " GPU(s)...\n";
    std::cout << "Base seed: 0x" << std::hex << base_seed << std::dec << "\n";

    if (config.benchmark_mode) {
        std::cout << "Running in benchmark mode for " << config.benchmark_seconds << " seconds...\n";
    }
    std::cout << "\n";

    // Launch GPU worker threads
    std::vector<std::thread> workers;
    for (auto &gpu: gpus) {
        workers.emplace_back(gpu_worker, gpu.get(), &results, base_seed);
    }

    // Optionally set main thread to high priority
#ifdef _WIN32
    SetThreadPriority(GetCurrentThread(), THREAD_PRIORITY_ABOVE_NORMAL);
#else
    struct sched_param param;
    param.sched_priority = 1;
    pthread_setschedparam(pthread_self(), SCHED_RR, &param);
#endif

    // Monitor progress
    if (config.benchmark_mode) {
        auto benchmark_end = std::chrono::steady_clock::now() +
                             std::chrono::seconds(config.benchmark_seconds);

        while (!g_shutdown.load() && std::chrono::steady_clock::now() < benchmark_end) {
            std::this_thread::sleep_for(std::chrono::seconds(1));
            monitor.update(gpus);
        }
        g_shutdown.store(true);
    } else {
        while (!g_shutdown.load()) {
            std::this_thread::sleep_for(std::chrono::seconds(1));
            monitor.update(gpus);
        }
    }

    // Wait for workers to finish
    for (auto &worker: workers) {
        worker.join();
    }

    // Final report
    monitor.finalReport();

    // Additional benchmark statistics
    if (config.benchmark_mode) {
        auto total_time = std::chrono::steady_clock::now() - g_start_time;
        double seconds = std::chrono::duration<double>(total_time).count();
        uint64_t total_hashes = g_total_hashes.load();

        std::cout << "\n=== Benchmark Results ===\n";
        std::cout << "Total GPUs    : " << gpus.size() << "\n";
        std::cout << "Duration      : " << std::fixed << std::setprecision(2) << seconds << " seconds\n";
        std::cout << "Total Hashes  : " << total_hashes / 1e12 << " trillion\n";
        std::cout << "Performance   : " << total_hashes / seconds / 1e9 << " GH/s\n";
        std::cout << "Per GPU       : " << total_hashes / seconds / 1e9 / gpus.size() << " GH/s\n";

        // Detailed per-GPU stats
        std::cout << "\nPer-GPU Performance:\n";
        for (size_t i = 0; i < gpus.size(); i++) {
            uint64_t gpu_hashes = gpus[i]->hashes_processed.load();
            double gpu_ghps = gpu_hashes / seconds / 1e9;
            uint64_t gpu_kernels = gpus[i]->kernels_launched.load();

            std::cout << "  GPU " << gpus[i]->device_id << " ("
                    << gpus[i]->properties.name << ", "
                    << getKernelName(gpus[i]->kernel_type) << "):\n";
            std::cout << "    Performance: " << gpu_ghps << " GH/s\n";
            std::cout << "    Kernels launched: " << gpu_kernels << "\n";
            std::cout << "    Avg kernel time: " << (seconds * 1000.0) / gpu_kernels << " ms\n";
        }
    }

    if (g_total_candidates.load() > 0) {
        std::cout << "\n[SUCCESS] Found " << g_total_candidates.load() << " collisions!\n";
        std::cout << "Results saved to: " << config.output_file << "\n";
    } else if (!config.benchmark_mode) {
        std::cout << "\n[CONTINUING] No collisions found yet. Keep mining!\n";
    }

    return 0;
}
