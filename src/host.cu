#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <array>
#include <vector>
#include <chrono>
#include <thread>
#include <atomic>
#include <csignal>
#include <fstream>
#include <sstream>
#include <ctime>
#include <mutex>
#include <memory>
#include <algorithm>
#include <random>
#include <cstring>

#ifdef _WIN32
#include <windows.h>
#else
    #include <pthread.h>
    #include <sched.h>
#endif

#include "job_upload_api.h"
#include "cxxsha1.hpp"

// Kernel declarations - updated to match our fixed kernels
extern "C" __global__ void sha1_mining_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_warp_collaborative_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_vectorized_kernel(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_bitsliced_kernel_correct(uint64_t *, uint32_t *, uint64_t);

extern "C" __global__ void sha1_hashcat_kernel(uint64_t *, uint32_t *, uint64_t);

#define CUDA_CHECK(e) do{ hipError_t _e=(e); \
    if(_e!=hipSuccess){ \
        std::cerr << "CUDA Error: " << hipGetErrorString(_e) \
                  << " at " << __FILE__ << ":" << __LINE__ << '\n'; \
        std::exit(1);} \
    }while(0)

// ==================== Configuration ====================
constexpr uint32_t RING_SIZE = 1u << 20; // 1M candidate slots
constexpr int PROGRESS_INTERVAL = 5; // Update every 5 seconds
constexpr int STREAMS_PER_GPU = 8; // More streams for better overlap

// Kernel types
enum KernelType {
    KERNEL_STANDARD,
    KERNEL_WARP_COLLABORATIVE,
    KERNEL_VECTORIZED,
    KERNEL_BITSLICED,
    KERNEL_HASHCAT,
    KERNEL_HASHCAT_EXTREME
};

// Global state for signal handling
std::atomic<bool> g_shutdown(false);
std::atomic<uint64_t> g_total_hashes(0);
std::atomic<uint32_t> g_total_candidates(0);
std::chrono::steady_clock::time_point g_start_time;

// Signal handler for graceful shutdown
void signal_handler(int sig) {
    std::cout << "\n\nReceived signal " << sig << ", shutting down gracefully...\n";
    g_shutdown.store(true);
}

void setup_signal_handlers() {
    std::signal(SIGINT, signal_handler);
    std::signal(SIGTERM, signal_handler);
#ifdef _WIN32
    std::signal(SIGBREAK, signal_handler);
#endif
}

// ==================== GPU Configuration ====================
struct GPUContext {
    int device_id;
    hipDeviceProp_t properties;
    std::vector<hipStream_t> streams;
    std::vector<uint64_t *> d_pairs;
    std::vector<uint32_t *> d_tickets;
    int optimal_blocks;
    int optimal_threads;
    uint64_t hashes_processed;
    KernelType kernel_type;
    int warps_per_block;
    uint64_t hashes_per_kernel_launch;

    GPUContext(int id, bool force_bitsliced = false, bool use_hashcat = true) : device_id(id), hashes_processed(0) {
        CUDA_CHECK(hipSetDevice(device_id));
        CUDA_CHECK(hipGetDeviceProperties(&properties, device_id));

        // Determine optimal configuration based on GPU
        if (force_bitsliced) {
            // Force bitsliced kernel for testing
            kernel_type = KERNEL_BITSLICED;
            optimal_threads = 128; // 4 warps, must be multiple of 32
            warps_per_block = 4;
            optimal_blocks = properties.multiProcessorCount * 8;
            // Bitsliced: each warp processes 32 messages * 4 batches = 128 messages
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 128;
        } else if (use_hashcat) {
            // Use HashCat-style kernels for maximum performance
            if (properties.major >= 8) {
                // Ampere and newer - use extreme HashCat kernel
                kernel_type = KERNEL_HASHCAT;
                optimal_threads = 256;
                warps_per_block = 8; // 128 threads = 4 warps
                optimal_blocks = properties.multiProcessorCount * 32 * 100;
                // Each thread processes 32 hashes
                hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 64;
            } else {
                // Older GPUs - use standard HashCat kernel
                kernel_type = KERNEL_HASHCAT;
                optimal_threads = 64;
                warps_per_block = 2; // 64 threads = 2 warps
                optimal_blocks = properties.multiProcessorCount * 64;
                // Each thread processes 16 hashes
                hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 16;
            }
        } else if (properties.major >= 7) {
            // Volta and newer - use warp collaborative
            kernel_type = KERNEL_WARP_COLLABORATIVE;
            optimal_threads = 256;
            warps_per_block = 8;
            optimal_blocks = properties.multiProcessorCount * 32;
            // Each warp processes 8 batches of 32 = 256 messages
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * warps_per_block * 256;
        } else if (properties.major >= 6) {
            // Pascal - use vectorized
            kernel_type = KERNEL_VECTORIZED;
            optimal_threads = 128;
            warps_per_block = 4;
            optimal_blocks = properties.multiProcessorCount * 16;
            // Each thread processes 2 messages
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 2;
        } else {
            // Older GPUs - use standard
            kernel_type = KERNEL_STANDARD;
            optimal_threads = 256;
            warps_per_block = 8;
            optimal_blocks = properties.multiProcessorCount * 8;
            // Each thread processes 4 messages
            hashes_per_kernel_launch = (uint64_t) optimal_blocks * optimal_threads * 4;
        }

        // Check available memory
        size_t free_mem, total_mem;
        CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));

        // Reserve some memory for system
        free_mem = (size_t) (free_mem * 0.9);

        size_t mem_per_stream = sizeof(uint64_t) * 4 * RING_SIZE + sizeof(uint32_t);
        int num_streams = std::min(STREAMS_PER_GPU, (int) (free_mem / mem_per_stream));

        // Create streams and allocate memory
        streams.resize(num_streams);
        d_pairs.resize(num_streams);
        d_tickets.resize(num_streams);

        for (int i = 0; i < num_streams; i++) {
            CUDA_CHECK(hipStreamCreateWithPriority(&streams[i], hipStreamNonBlocking,
                i % 2)); // Alternate priorities
            CUDA_CHECK(hipMalloc(&d_pairs[i], sizeof(uint64_t) * 4 * RING_SIZE));
            CUDA_CHECK(hipMalloc(&d_tickets[i], sizeof(uint32_t)));
            CUDA_CHECK(hipMemsetAsync(d_tickets[i], 0, sizeof(uint32_t), streams[i]));
        }

        // Set cache configuration based on kernel
        switch (kernel_type) {
            case KERNEL_STANDARD:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_mining_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_WARP_COLLABORATIVE:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_warp_collaborative_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_VECTORIZED:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_vectorized_kernel), hipFuncCachePreferShared));
                break;
            case KERNEL_BITSLICED:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_bitsliced_kernel_correct), hipFuncCachePreferL1));
                break;
            case KERNEL_HASHCAT:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_hashcat_kernel), hipFuncCachePreferL1));
                break;
            case KERNEL_HASHCAT_EXTREME:
                CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(sha1_warp_collaborative_kernel), hipFuncCachePreferL1));
                break;
        }

        printInfo();
    }

    ~GPUContext() {
        CUDA_CHECK(hipSetDevice(device_id));
        for (size_t i = 0; i < streams.size(); i++) {
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
            CUDA_CHECK(hipStreamDestroy(streams[i]));
            CUDA_CHECK(hipFree(d_pairs[i]));
            CUDA_CHECK(hipFree(d_tickets[i]));
        }
    }

    void printInfo() {
        std::cout << "=== GPU " << device_id << " Configuration ===\n";
        std::cout << "Device        : " << properties.name << "\n";
        std::cout << "Compute Cap   : " << properties.major << "." << properties.minor << "\n";
        std::cout << "SMs           : " << properties.multiProcessorCount << "\n";
        std::cout << "Clock Rate    : " << properties.clockRate / 1000 << " MHz\n";
        std::cout << "Memory        : " << properties.totalGlobalMem / (1024.0 * 1024 * 1024) << " GB\n";
        std::cout << "Memory Clock  : " << properties.memoryClockRate / 1000 << " MHz\n";
        std::cout << "Memory Bus    : " << properties.memoryBusWidth << " bits\n";
        std::cout << "L2 Cache      : " << properties.l2CacheSize / (1024 * 1024) << " MB\n";

        const char *kernel_names[] = {
            "Standard", "Warp Collaborative", "Vectorized", "Bitsliced", "HashCat", "HashCat Extreme"
        };
        std::cout << "Kernel        : " << kernel_names[kernel_type] << "\n";
        std::cout << "Configuration : " << optimal_blocks << " blocks × " << optimal_threads << " threads\n";
        if (kernel_type == KERNEL_HASHCAT || kernel_type == KERNEL_HASHCAT_EXTREME) {
            int hashes_per_thread = (kernel_type == KERNEL_HASHCAT_EXTREME) ? 32 : 16;
            std::cout << "Hashes/Thread : " << hashes_per_thread << "\n";
        } else {
            std::cout << "Warps/Block   : " << warps_per_block << "\n";
        }
        std::cout << "Hashes/Launch : " << hashes_per_kernel_launch << "\n";
        std::cout << "Streams       : " << streams.size() << "\n\n";
    }

    void launchKernel(hipStream_t stream, uint64_t seed) {
        switch (kernel_type) {
            case KERNEL_STANDARD:
                sha1_mining_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[getStreamIndex(stream)],
                    d_tickets[getStreamIndex(stream)],
                    seed
                );
                break;
            case KERNEL_WARP_COLLABORATIVE:
                sha1_warp_collaborative_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[getStreamIndex(stream)],
                    d_tickets[getStreamIndex(stream)],
                    seed
                );
                break;
            case KERNEL_VECTORIZED:
                sha1_vectorized_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[getStreamIndex(stream)],
                    d_tickets[getStreamIndex(stream)],
                    seed
                );
                break;
            case KERNEL_BITSLICED:
                // Bitsliced kernel requires thread count to be multiple of 32
                sha1_bitsliced_kernel_correct<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[getStreamIndex(stream)],
                    d_tickets[getStreamIndex(stream)],
                    seed
                );
                break;
            case KERNEL_HASHCAT:
                sha1_hashcat_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[getStreamIndex(stream)],
                    d_tickets[getStreamIndex(stream)],
                    seed
                );
                break;
            case KERNEL_HASHCAT_EXTREME:
                sha1_warp_collaborative_kernel<<<optimal_blocks, optimal_threads, 0, stream>>>(
                    d_pairs[getStreamIndex(stream)],
                    d_tickets[getStreamIndex(stream)],
                    seed
                );
                break;
        }
    }

private:
    int getStreamIndex(hipStream_t stream) {
        for (size_t i = 0; i < streams.size(); i++) {
            if (streams[i] == stream) return i;
        }
        return 0;
    }
};

// ==================== Result Handler ====================
class ResultHandler {
private:
    std::ofstream output_file;
    std::mutex file_mutex;
    std::array<uint8_t, 20> target_hash;
    std::array<uint8_t, 32> base_message;

public:
    ResultHandler(const std::string &filename, const uint8_t target[20], const uint8_t base_msg[32]) {
        std::copy(target, target + 20, target_hash.begin());
        std::copy(base_msg, base_msg + 32, base_message.begin());

        output_file.open(filename, std::ios::app);
        if (!output_file.is_open()) {
            std::cerr << "Failed to open output file: " << filename << "\n";
        }

        auto now = std::chrono::system_clock::now();
        auto time_t = std::chrono::system_clock::to_time_t(now);
        output_file << "\n=== SHA-1 Collision Mining Started at "
                << std::put_time(std::localtime(&time_t), "%Y-%m-%d %H:%M:%S")
                << " ===\n";
        output_file << "Target: ";
        for (int i = 0; i < 20; i++) {
            output_file << std::hex << std::setw(2) << std::setfill('0') << (int) target_hash[i];
        }
        output_file << "\n\n";
        output_file.flush();
    }

    void saveResults(uint64_t *h_pairs, uint32_t count, int gpu_id) {
        std::lock_guard<std::mutex> lock(file_mutex);

        output_file << "[GPU " << gpu_id << "] Found " << count << " collision candidates:\n";

        for (uint32_t i = 0; i < std::min(count, 100u); i++) {
            output_file << "Candidate " << i + 1 << ":\n";

            // Reconstruct message
            uint8_t msg[32];
            for (int j = 0; j < 4; j++) {
                uint64_t word = h_pairs[i * 4 + j];
                for (int k = 0; k < 8; k++) {
                    msg[j * 8 + k] = (word >> (k * 8)) & 0xFF;
                }
            }

            output_file << "  Message: ";
            for (int j = 0; j < 32; j++) {
                output_file << std::hex << std::setw(2) << std::setfill('0') << (int) msg[j];
            }
            output_file << "\n";

            // Verify SHA-1
            uint8_t hash[20];
            sha1_ctx ctx;
            sha1_init(ctx);
            sha1_update(ctx, msg, 32);
            sha1_final(ctx, hash);

            output_file << "  SHA-1:   ";
            for (int j = 0; j < 20; j++) {
                output_file << std::hex << std::setw(2) << std::setfill('0') << (int) hash[j];
            }

            bool verified = (std::memcmp(hash, target_hash.data(), 20) == 0);
            output_file << " [" << (verified ? "VERIFIED" : "FAILED") << "]\n\n";
        }

        output_file.flush();

        // Console notification
        std::cout << "\n[GPU " << gpu_id << "] Found " << count << " collision candidates! ";
        std::cout << "Saved to output file.\n";
    }
};

// ==================== Performance Monitor ====================
class PerformanceMonitor {
private:
    std::chrono::steady_clock::time_point last_update;
    uint64_t last_hashes = 0;
    std::mutex monitor_mutex;
    std::vector<double> rate_history;
    const size_t history_size = 12; // 1 minute of 5-second intervals

public:
    PerformanceMonitor() {
        last_update = std::chrono::steady_clock::now();
        rate_history.reserve(history_size);
    }

    void update(const std::vector<std::unique_ptr<GPUContext> > &gpus) {
        auto now = std::chrono::steady_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - last_update);

        if (elapsed.count() >= PROGRESS_INTERVAL) {
            std::lock_guard<std::mutex> lock(monitor_mutex);

            auto total_elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - g_start_time);
            double total_seconds = total_elapsed.count();

            uint64_t current_hashes = g_total_hashes.load();
            uint64_t interval_hashes = current_hashes - last_hashes;
            double interval_rate = interval_hashes / elapsed.count() / 1e9;
            double average_rate = current_hashes / total_seconds / 1e9;

            // Update rate history
            rate_history.push_back(interval_rate);
            if (rate_history.size() > history_size) {
                rate_history.erase(rate_history.begin());
            }

            // Calculate moving average
            double moving_avg = 0;
            for (double rate: rate_history) {
                moving_avg += rate;
            }
            moving_avg /= rate_history.size();

            std::cout << "\r[" << formatTime(total_seconds) << "] "
                    << "Hashes: " << std::fixed << std::setprecision(2)
                    << current_hashes / 1e12 << "T | "
                    << "Rate: " << interval_rate << " GH/s (avg: "
                    << average_rate << " GH/s, MA: " << moving_avg << ") | "
                    << "Candidates: " << g_total_candidates.load();

            // Per-GPU stats
            std::cout << " | GPU: ";
            for (size_t i = 0; i < gpus.size(); i++) {
                double gpu_rate = gpus[i]->hashes_processed / total_seconds / 1e9;
                std::cout << "[" << i << ":" << std::fixed << std::setprecision(1)
                        << gpu_rate << "] ";
            }
            std::cout << "    " << std::flush;

            last_update = now;
            last_hashes = current_hashes;
        }
    }

    void finalReport() {
        auto now = std::chrono::steady_clock::now();
        auto total_elapsed = std::chrono::duration_cast<std::chrono::seconds>(now - g_start_time);
        double total_seconds = total_elapsed.count();
        uint64_t total_hashes = g_total_hashes.load();
        double average_rate = total_hashes / total_seconds / 1e9;

        std::cout << "\n\n=== Final Statistics ===\n";
        std::cout << "Total Time    : " << formatTime(total_seconds) << "\n";
        std::cout << "Total Hashes  : " << std::fixed << std::setprecision(3)
                << total_hashes / 1e12 << " trillion\n";
        std::cout << "Average Rate  : " << average_rate << " GH/s\n";
        std::cout << "Candidates    : " << g_total_candidates.load() << "\n";
        std::cout << "Efficiency    : " << std::scientific << std::setprecision(2)
                << (double) g_total_candidates.load() / total_hashes * 100 << "%\n";
    }

private:
    std::string formatTime(double seconds) {
        int hours = seconds / 3600;
        int minutes = (seconds - hours * 3600) / 60;
        int secs = seconds - hours * 3600 - minutes * 60;

        std::stringstream ss;
        ss << std::setfill('0') << std::setw(2) << hours << ":"
                << std::setw(2) << minutes << ":"
                << std::setw(2) << secs;
        return ss.str();
    }
};

// ==================== GPU Worker Thread ====================
void gpu_worker(GPUContext *gpu, ResultHandler *results, uint64_t base_seed) {
    CUDA_CHECK(hipSetDevice(gpu->device_id));

    uint64_t local_seed = base_seed + gpu->device_id * (1ull << 48);

    // Allocate pinned host memory for faster transfers
    std::vector<uint64_t> h_pairs;
    h_pairs.resize(RING_SIZE * 4);

    while (!g_shutdown.load()) {
        // Launch kernels on all streams
        for (size_t s = 0; s < gpu->streams.size(); s++) {
            gpu->launchKernel(gpu->streams[s], local_seed);
            local_seed += gpu->hashes_per_kernel_launch;
        }

        // Check results from completed streams
        for (size_t s = 0; s < gpu->streams.size(); s++) {
            // Use events for more efficient synchronization
            hipError_t err = hipStreamQuery(gpu->streams[s]);
            if (err == hipSuccess) {
                uint32_t found = 0;
                CUDA_CHECK(hipMemcpyAsync(&found, gpu->d_tickets[s], sizeof(uint32_t),
                    hipMemcpyDeviceToHost, gpu->streams[s]));
                CUDA_CHECK(hipStreamSynchronize(gpu->streams[s]));

                if (found > 0) {
                    g_total_candidates.fetch_add(found);

                    uint32_t to_copy = std::min(found, RING_SIZE);
                    CUDA_CHECK(hipMemcpyAsync(h_pairs.data(), gpu->d_pairs[s],
                        sizeof(uint64_t) * 4 * to_copy, hipMemcpyDeviceToHost, gpu->streams[s]));
                    CUDA_CHECK(hipStreamSynchronize(gpu->streams[s]));

                    results->saveResults(h_pairs.data(), to_copy, gpu->device_id);

                    // Reset ticket
                    CUDA_CHECK(hipMemsetAsync(gpu->d_tickets[s], 0, sizeof(uint32_t), gpu->streams[s]));
                }

                // Update counters
                g_total_hashes.fetch_add(gpu->hashes_per_kernel_launch);
                gpu->hashes_processed += gpu->hashes_per_kernel_launch;
            } else if (err != hipErrorNotReady) {
                CUDA_CHECK(err);
            }
        }

        // Small delay to prevent CPU spinning
        std::this_thread::sleep_for(std::chrono::microseconds(100));
    }

    // Ensure all streams finish
    for (auto &stream: gpu->streams) {
        CUDA_CHECK(hipStreamSynchronize(stream));
    }
}

// ==================== Configuration Parser ====================
struct Config {
    std::vector<int> gpu_ids;
    std::string output_file = "sha1_collisions.txt";
    std::array<uint8_t, 32> target_preimage;
    bool benchmark_mode = false;
    int benchmark_seconds = 60;
    bool force_bitsliced = false;
    bool no_hashcat = false;
};

Config parse_arguments(int argc, char **argv) {
    Config config;

    // Initialize with zeros
    config.target_preimage.fill(0);

    for (int i = 1; i < argc; i++) {
        std::string arg = argv[i];

        if (arg == "--gpu" && i + 1 < argc) {
            config.gpu_ids.push_back(std::stoi(argv[++i]));
        } else if (arg == "--output" && i + 1 < argc) {
            config.output_file = argv[++i];
        } else if (arg == "--target" && i + 1 < argc) {
            std::string hex = argv[++i];
            if (hex.length() != 64) {
                std::cerr << "Error: Target must be 64 hex characters (32 bytes)\n";
                std::exit(1);
            }
            for (int j = 0; j < 32; j++) {
                std::string byte = hex.substr(j * 2, 2);
                config.target_preimage[j] = std::stoi(byte, nullptr, 16);
            }
        } else if (arg == "--benchmark") {
            config.benchmark_mode = true;
            if (i + 1 < argc && argv[i + 1][0] != '-') {
                config.benchmark_seconds = std::stoi(argv[++i]);
            }
        } else if (arg == "--bitsliced") {
            config.force_bitsliced = true;
        } else if (arg == "--no-hashcat") {
            config.no_hashcat = true;
        } else if (arg == "--help") {
            std::cout << "SHA-1 Collision Miner v3.0\n\n";
            std::cout << "Usage: " << argv[0] << " [options]\n\n";
            std::cout << "Options:\n";
            std::cout << "  --gpu <id>       GPU device ID (can be used multiple times)\n";
            std::cout << "  --output <file>  Output file (default: sha1_collisions.txt)\n";
            std::cout << "  --target <hex>   Target preimage in hex (default: zeros)\n";
            std::cout << "  --benchmark [s]  Run benchmark for s seconds (default: 60)\n";
            std::cout << "  --bitsliced      Force bitsliced kernel (slower, for testing)\n";
            std::cout << "  --no-hashcat     Use original kernels instead of HashCat-style\n";
            std::cout << "  --help           Show this help\n\n";
            std::cout << "Example:\n";
            std::cout << "  " << argv[0] << " --gpu 0 --gpu 1 --target ";
            std::cout << "0123456789abcdef0123456789abcdef0123456789abcdef0123456789abcdef\n";
            std::exit(0);
        } else {
            std::cerr << "Unknown option: " << arg << "\n";
            std::cerr << "Use --help for usage information\n";
            std::exit(1);
        }
    }

    // Default: use all available GPUs
    if (config.gpu_ids.empty()) {
        int num_gpus;
        CUDA_CHECK(hipGetDeviceCount(&num_gpus));
        for (int i = 0; i < num_gpus; i++) {
            config.gpu_ids.push_back(i);
        }
    }

    return config;
}

// ==================== Main Function ====================
int main(int argc, char **argv) {
    setup_signal_handlers();

    std::cout << "\n+------------------------------------------+\n";
    std::cout << "|      SHA-1 Collision Miner v3.0         |\n";
    std::cout << "+------------------------------------------+\n\n";

    // Parse configuration
    Config config = parse_arguments(argc, argv);

    // Calculate target SHA-1
    uint8_t target_hash[20];
    sha1_ctx ctx;
    sha1_init(ctx);
    sha1_update(ctx, config.target_preimage.data(), 32);
    sha1_final(ctx, target_hash);

    // Convert to uint32_t array for GPU
    uint32_t target[5];
    for (int i = 0; i < 5; i++) {
        target[i] = (uint32_t(target_hash[4 * i]) << 24) |
                    (uint32_t(target_hash[4 * i + 1]) << 16) |
                    (uint32_t(target_hash[4 * i + 2]) << 8) |
                    uint32_t(target_hash[4 * i + 3]);
    }

    // Upload job to GPU constant memory
    upload_new_job(config.target_preimage.data(), target);

    std::cout << "Target Preimage: ";
    for (int i = 0; i < 32; i++) {
        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int) config.target_preimage[i];
    }
    std::cout << "\n";

    std::cout << "Target SHA-1: ";
    for (int i = 0; i < 20; i++) {
        std::cout << std::hex << std::setw(2) << std::setfill('0') << (int) target_hash[i];
    }
    std::cout << std::dec << "\n\n";

    if (config.force_bitsliced) {
        std::cout << "NOTE: Using bitsliced kernel (slower, for testing)\n\n";
    }

    // Initialize GPUs
    std::vector<std::unique_ptr<GPUContext> > gpus;
    for (int id: config.gpu_ids) {
        try {
            gpus.push_back(std::make_unique<GPUContext>(id, config.force_bitsliced, !config.no_hashcat));
        } catch (const std::exception &e) {
            std::cerr << "Failed to initialize GPU " << id << ": " << e.what() << "\n";
        }
    }

    if (gpus.empty()) {
        std::cerr << "No GPUs available for mining!\n";
        return 1;
    }

    // Calculate expected total performance
    uint64_t total_hashes_per_second = 0;
    for (const auto &gpu: gpus) {
        // Estimate based on kernel type and GPU
        uint64_t gpu_hps;
        if (gpu->kernel_type == KERNEL_BITSLICED) {
            // Bitsliced is much slower
            gpu_hps = gpu->hashes_per_kernel_launch * gpu->streams.size() * 100; // ~10ms per kernel
        } else if (gpu->kernel_type == KERNEL_HASHCAT || gpu->kernel_type == KERNEL_HASHCAT_EXTREME) {
            // HashCat kernels are highly optimized
            gpu_hps = gpu->hashes_per_kernel_launch * gpu->streams.size() * 2000; // ~0.5ms per kernel
        } else {
            gpu_hps = gpu->hashes_per_kernel_launch * gpu->streams.size() * 1000; // ~1ms per kernel
        }
        total_hashes_per_second += gpu_hps;
    }
    std::cout << "Expected performance: ~" << total_hashes_per_second / 1e9 << " GH/s\n\n";

    // Initialize result handler
    ResultHandler results(config.output_file, target_hash, config.target_preimage.data());

    // Initialize performance monitor
    PerformanceMonitor monitor;

    // Start timing
    g_start_time = std::chrono::steady_clock::now();

    // Generate random base seed
    std::random_device rd;
    uint64_t base_seed = (uint64_t(rd()) << 32) | rd();

    std::cout << "Starting collision search with " << gpus.size() << " GPU(s)...\n";
    std::cout << "Base seed: 0x" << std::hex << base_seed << std::dec << "\n";

    if (config.benchmark_mode) {
        std::cout << "Running in benchmark mode for " << config.benchmark_seconds << " seconds...\n";
    }
    std::cout << "\n";

    // Launch GPU worker threads
    std::vector<std::thread> workers;
    for (auto &gpu: gpus) {
        workers.emplace_back(gpu_worker, gpu.get(), &results, base_seed);
    }

    // Optionally set main thread to high priority
#ifdef _WIN32
    SetThreadPriority(GetCurrentThread(), THREAD_PRIORITY_ABOVE_NORMAL);
#else
    struct sched_param param;
    param.sched_priority = 1;
    pthread_setschedparam(pthread_self(), SCHED_RR, &param);
#endif

    // Monitor progress
    if (config.benchmark_mode) {
        auto benchmark_end = std::chrono::steady_clock::now() +
                             std::chrono::seconds(config.benchmark_seconds);

        while (!g_shutdown.load() && std::chrono::steady_clock::now() < benchmark_end) {
            std::this_thread::sleep_for(std::chrono::seconds(1));
            monitor.update(gpus);
        }
        g_shutdown.store(true);
    } else {
        while (!g_shutdown.load()) {
            std::this_thread::sleep_for(std::chrono::seconds(1));
            monitor.update(gpus);
        }
    }

    // Wait for workers to finish
    for (auto &worker: workers) {
        worker.join();
    }

    // Final report
    monitor.finalReport();

    // Additional benchmark statistics
    if (config.benchmark_mode) {
        auto total_time = std::chrono::steady_clock::now() - g_start_time;
        double seconds = std::chrono::duration<double>(total_time).count();
        uint64_t total_hashes = g_total_hashes.load();

        std::cout << "\n=== Benchmark Results ===\n";
        std::cout << "Total GPUs    : " << gpus.size() << "\n";
        std::cout << "Duration      : " << std::fixed << std::setprecision(2) << seconds << " seconds\n";
        std::cout << "Total Hashes  : " << total_hashes / 1e12 << " trillion\n";
        std::cout << "Performance   : " << total_hashes / seconds / 1e9 << " GH/s\n";
        std::cout << "Per GPU       : " << total_hashes / seconds / 1e9 / gpus.size() << " GH/s\n";

        // Detailed per-GPU stats
        std::cout << "\nPer-GPU Performance:\n";
        for (size_t i = 0; i < gpus.size(); i++) {
            double gpu_ghps = gpus[i]->hashes_processed / seconds / 1e9;
            const char *kernel_names[] = {
                "Standard", "Warp Collaborative", "Vectorized", "Bitsliced", "HashCat", "HashCat Extreme"
            };
            std::cout << "  GPU " << gpus[i]->device_id << " ("
                    << gpus[i]->properties.name << ", "
                    << kernel_names[gpus[i]->kernel_type] << "): "
                    << gpu_ghps << " GH/s\n";
        }
    }

    if (g_total_candidates.load() > 0) {
        std::cout << "\n[SUCCESS] Found " << g_total_candidates.load() << " collisions!\n";
        std::cout << "Results saved to: " << config.output_file << "\n";
    } else if (!config.benchmark_mode) {
        std::cout << "\n[CONTINUING] No collisions found yet. Keep mining!\n";
    }

    return 0;
}
